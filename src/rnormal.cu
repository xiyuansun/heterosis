#include "hip/hip_runtime.h"
#include <constants.h>
#include <functions.h>
#include <math.h>
#include <stdlib.h>

num_t rnormal(num_t m, num_t s){

  num_t u1 = runiform(0, 1);
  num_t u2 = runiform(0, 1);
  
  return sqrt(-2 * log(u1)) * sin(2 * M_PI * u2) * s + m;
}