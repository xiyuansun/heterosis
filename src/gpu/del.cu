#include "hip/hip_runtime.h"
#include <Chain.h>
#include <constants.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

num_t delProp(Chain *a, int g){ /* device */
  int G = a->G;      

  num_t gam = a->gamDel;
  num_t sig = a->sigDel[a->mSigDel];

  num_t gprec = 1/(gam * gam);
  num_t sprec = 1/(sig * sig);

  num_t avg = (a->del[iG(a->mDel, g)] * sprec) / (gprec + sprec);
  num_t s = gam * gam + sig * sig;
  num_t u = runiform(0, 1);
  num_t nw;

  if(u < a->piDel[a->mPiDel]){
    nw = 0;
  } else {
    nw = rnormal(avg, s);
  }

  return nw;
}

num_t lDel(Chain *a, int g, num_t arg){ /* device */ 
  int n, N = a->N, G = a->G;
  num_t s = 0, tmp; 
  
  for(n = 0; n < a->N; ++n){
    if(a->grp[n] != 2){
      tmp = mu(a, n, a->phi[iG(a->mPhi, g)], a->alp[iG(a->mAlp, g)], arg);
      s += a->y[iG(n, g)] * tmp - exp(a->c[iN(a->mC, n)] + 
          a->eps[iNG(a->mEps, n, g)] + tmp);
    }
  }
 
  if(arg * arg > 1e-6){
    tmp = -pow(arg - a->theDel[a->mTheDel], 2) / (2 * pow(a->sigDel[a->mSigDel], 2)) -
                log(1 - a->piDel[a->mPiDel]);
  } else {
    tmp = log(a->piDel[a->mPiDel]);
  }

  return s + tmp;
}

void sampleDel_kernel1(Chain *a){ /* kernel <<<G, 1>>> */
  int g, G = a->G;
  num_t old, nw, dl, lp, lu;

  for(g = 0; g < a->G; ++g){ 

    old = a->del[iG(a->mDel, g)];
    nw = delProp(a, g);
    
    dl = lDel(a, g, nw) - lDel(a, g, old);
    lp = 0 < dl? 0 : dl;
    lu = log(runiform(0, 1));
    
    if(lu < lp){ /* accept */
      a->del[iG(a->mDel + 1, g)] = nw;
      
      if(a->mDel >= a->burnin)
        ++a->accDel[g];
    } else { /* reject */
      a->del[iG(a->mDel + 1, g)] = old;
    }
  }
}

void sampleDel_kernel2(Chain *a){ /* kernel <<<1 1>>> */
  ++a->mDel;
}

void sampleDel(Chain *a){ /* host */
  sampleDel_kernel1(a);
  sampleDel_kernel2(a);
}