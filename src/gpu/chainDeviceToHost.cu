#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <functions.h>
#include <stdio.h>
#include <stdlib.h>

__host__ Chain *chainDeviceToHost(Chain *a, Config *cfg){

  Chain *host_a = allocChain(cfg, 1); 
  
  /* program options */
  
  CUDA_CALL(hipMemcpy(&(host_a->M), &(a->M), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->N), &(a->N), sizeof(int), hipMemcpyDeviceToHost)); 
  CUDA_CALL(hipMemcpy(&(host_a->G), &(a->G), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->burnin), &(a->burnin), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->heterosis), &(a->heterosis), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->someParmsFlag), &(a->someParmsFlag), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->allParmsFlag), &(a->allParmsFlag), sizeof(int), hipMemcpyDeviceToHost));
  
  /* data */
printf("2\n");
  CUDA_CALL(hipMemcpy(host_a->y, a->y, cfg->N * cfg->G * sizeof(count_t), hipMemcpyDeviceToHost)); printf("3\n"); 
  CUDA_CALL(hipMemcpy(host_a->yMeanG, a->yMeanG, cfg->N * sizeof(num_t), hipMemcpyDeviceToHost));printf("4\n");
  CUDA_CALL(hipMemcpy(host_a->grp, a->grp, cfg->N * sizeof(int), hipMemcpyDeviceToHost));printf("5\n");
  
  /* initialization constants */
  CUDA_CALL(hipMemcpy(&(host_a->sigC0), &(a->sigC0), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->d0), &(a->d0), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->aTau), &(a->aTau), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->aAlp), &(a->aAlp), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->aDel), &(a->aDel), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->bTau), &(a->bTau), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->bAlp), &(a->bAlp), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->bDel), &(a->bDel), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->gamPhi), &(a->gamPhi), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->gamAlp), &(a->gamAlp), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->gamDel), &(a->gamDel), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->sigPhi0), &(a->sigPhi0), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->sigAlp0), &(a->sigAlp0), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->sigDel0), &(a->sigDel0), sizeof(num_t), hipMemcpyDeviceToHost));
  
  /* parameters */
  
  CUDA_CALL(hipMemcpy(host_a->c, a->c, (cfg->M + 1) * cfg->N * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->sigC, a->sigC, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->eps, a->eps, (cfg->M + 1) * cfg->N * cfg->G * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->eta, a->eta, (cfg->M + 1) * cfg->G * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->d, a->d, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->tau, a->tau, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->phi, a->phi, (cfg->M + 1) * cfg->G * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->thePhi, a->thePhi, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->sigPhi, a->sigPhi, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->alp, a->alp, (cfg->M + 1) * cfg->G * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->theAlp, a->theAlp, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->sigAlp, a->sigAlp, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->piAlp, a->piAlp, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->del, a->del, (cfg->M + 1) * cfg->G * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->theDel, a->theDel, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->sigDel, a->sigDel, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->piDel, a->piDel, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
  
  /* temporary and return values */
  
  CUDA_CALL(hipMemcpy(&(host_a->s1), &(a->s1), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->s2), &(a->s2), sizeof(num_t), hipMemcpyDeviceToHost));
  
  CUDA_CALL(hipMemcpy(host_a->tmp1, a->tmp1, cfg->G * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->tmp2, a->tmp2, cfg->G * sizeof(num_t), hipMemcpyDeviceToHost));
  
  CUDA_CALL(hipMemcpy(host_a->Old, a->Old, cfg->N * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->New, a->New, cfg->N * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->lOld, a->lOld, cfg->N * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->lNew, a->lNew, cfg->N * sizeof(num_t), hipMemcpyDeviceToHost));
  
  /* current place in the chain of each parameter */
    
  CUDA_CALL(hipMemcpy(&(host_a->mC), &(a->mC), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mSigC), &(a->mSigC), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mEps), &(a->mEps), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mEta), &(a->mEta), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mD), &(a->mD), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mTau), &(a->mTau), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mPhi), &(a->mPhi), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mAlp), &(a->mAlp), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mDel), &(a->mDel), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mThePhi), &(a->mThePhi), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mTheAlp), &(a->mTheAlp), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mTheDel), &(a->mTheDel), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mSigPhi), &(a->mSigPhi), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mSigAlp), &(a->mSigAlp), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mSigDel), &(a->mSigDel), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mPiAlp), &(a->mPiAlp), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->mPiDel), &(a->mPiDel), sizeof(int), hipMemcpyDeviceToHost));
  
  /* tuning parameters for Metropolis steps */
  
  CUDA_CALL(hipMemcpy(&(host_a->tuneD), &(a->tuneD), sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->tuneC, a->tuneC, cfg->N * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->tunePhi, a->tunePhi, cfg->G * sizeof(num_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->tuneEps, a->tuneEps, cfg->N * cfg->G * sizeof(num_t), hipMemcpyDeviceToHost));
  
  /* number of acceptances for Metropolis steps */
  
  CUDA_CALL(hipMemcpy(&(host_a->accD), &(a->accD), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->accC, a->accC, cfg->N * sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->accPhi, a->accPhi, cfg->G * sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->accAlp, a->accAlp, cfg->G * sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->accDel, a->accDel, cfg->G * sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_a->accEps, a->accEps, cfg->G * sizeof(int), hipMemcpyDeviceToHost));
  
  /* choices to hold hyperparameters constant */
  
  CUDA_CALL(hipMemcpy(&(host_a->constSigC), &(a->constSigC), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->constD), &(a->constD), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->constTau), &(a->constTau), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->constThePhi), &(a->constThePhi), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->constTheAlp), &(a->constTheAlp), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->constTheDel), &(a->constTheDel), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->constSigPhi), &(a->constSigPhi), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->constSigAlp), &(a->constSigAlp), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->constSigDel), &(a->constSigDel), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->constPiAlp), &(a->constPiAlp), sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(&(host_a->constPiDel), &(a->constPiDel), sizeof(int), hipMemcpyDeviceToHost));
  
  return a;
}