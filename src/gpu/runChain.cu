#include "hip/hip_runtime.h"
#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void runChain(Chain *host_a, Chain *dev_a, Config *cfg){
  int m;
  double time;
  time_t p1, p2;
  
  fprintf(cfg->log, "  Running chain.\n");
  time(p1);
  
  for(m = 0; m < cfg->M; ++m){
    fprintf(cfg->log, "    iter %d | ", m);

    sampleC(host_a, dev_a, cfg);
    sampleTau(host_a, dev_a, cfg);
    samplePiAlp(host_a, dev_a, cfg);
    samplePiDel(host_a, dev_a, cfg);
    sampleD(host_a, dev_a, cfg);
    sampleThePhi(host_a, dev_a, cfg);
    sampleTheAlp(host_a, dev_a, cfg);
    sampleTheDel(host_a, dev_a, cfg);
    sampleSigC(host_a, dev_a, cfg);
    sampleSigPhi(host_a, dev_a, cfg);
    sampleSigAlp(host_a, dev_a, cfg);
    sampleSigDel(host_a, dev_a, cfg);
    sampleEta(host_a, dev_a, cfg);
    sampleEps(host_a, dev_a, cfg);
    samplePhiAlpDel(host_a, dev_a, cfg);

    fprintf(cfg->log, "\n");
  }
  
  time(p2);
  fprintf(cfg->time, "%0.3f ", difftime(p2, p1)/60.0);
} 