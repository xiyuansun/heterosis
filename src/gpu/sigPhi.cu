#include "hip/hip_runtime.h"
#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/reduce.h>

__global__ void sampleSigPhi_kernel1(Chain *a){ /* kernel <<<G, 1>>> */
  int g = IDX;

  if(g < a->G) 
    a->tmp1[g] = pow(a->phi[g] - a->thePhi, 2);
}

__global__ void sampleSigPhi_kernel2(Chain *a){ /* kernel <<<1, 1>>> */
  num_t rate = a->s1 / 2;
  num_t shape = (a->G - 1) / 2;
  num_t lb = 1/pow(a->sigPhi0, 2);

  if(shape >= 1 && rate > 0){
    a->sigPhi = 1/sqrt(rgammaDevice(a, 1, shape, rate, lb));
  } else {
    a->sigPhi = a->sigPhi;
  }
}

void sampleSigPhi(Chain *host_a, Chain *dev_a, Config *cfg){ /* host */

  float myTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  if(cfg->verbose)
    printf("sigPhi ");

  if(cfg->constSigPhi)
    return;

  sampleSigPhi_kernel1<<<G_GRID, G_BLOCK>>>(dev_a);
  
  thrust::device_ptr<num_t> tmp1(host_a->tmp1);  
  num_t s1 = thrust::reduce(tmp1, tmp1 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s1), &s1, sizeof(num_t), hipMemcpyHostToDevice));
  
  sampleSigPhi_kernel2<<<1, 1>>>(dev_a);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&myTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  cfg->timeSigPhi = myTime;
}