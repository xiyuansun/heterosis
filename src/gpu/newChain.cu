#include "hip/hip_runtime.h"
#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

__host__ int cmpfunc (const void *a, const void *b){
   return ( *(num_t*)a - *(num_t*)b );
}

__global__ void newChain_kernel1(Chain *a){ /* kernel <<<G, 1>>> */
  int n, g = GENE, N = a->N, G = a->G;
  num_t u;

  a->phi[iG(0, g)] = 1; /* rnormal(a->thePhi[0], a->sigPhi[0]);*/

  u = 0.5; /* runiform(0, 1) */;
  if(u < a->piAlp[0]){
    a->alp[iG(0, g)] = 0;
  } else {
    a->alp[iG(0, g)] = 1; /* rnormal(a->theAlp[0], a->sigAlp[0]); */
  }
    
  u = 0.5; /* runiform(0, 1); */
  if(u < a->piDel[0]){
    a->del[iG(0, g)] = 0;
  } else {
    a->del[iG(0, g)] = 1; /* rnormal(a->theDel[0], a->sigDel[0]);*/
  }
 
  a->eta[iG(0, g)] = 1; /* 1/sqrt(rgamma(a->d[0] / 2, 
                 a->d[0] * a->tau[0] * a->tau[0] / 2, 0)); */

  for(n = 0; n < a->N; ++n)
    a->eps[iNG(0, n, g)] = 1; /* rnormal(0, a->eta[iG(0, g)]); */
    
}

__global__ void newChain_kernel2(Chain *a){ /* kernel <<<1, 1>>> */
  int n, g, G = a->G;

  a->mC = 0;
  a->mSigC = 0;

  a->mEps = 0;
  a->mEta = 0;
  a->mD = 0;
  a->mTau = 0;

  a->mPhi = 0;
  a->mAlp = 0;
  a->mDel = 0;

  a->mThePhi = 0;
  a->mTheAlp = 0;
  a->mTheDel = 0;

  a->mSigPhi = 0;
  a->mSigAlp = 0;
  a->mSigDel = 0;

  a->mPiAlp = 0;
  a->mPiDel = 0;

  a->tuneD = 100;

  for(n = 0; n < a->N; ++n)
    a->tuneC[n] = 1;

  for(g = 0; g < a->G; ++g){
    a->tunePhi[g] = 1;

    for(n = 0; n < a->N; ++n)
      a->tuneEps[iG(n, g)] = 1;
  }
  
  a->accD = 0;

  for(n = 0; n < a->N; ++n){
    a->accC[n] = 0;
  
    for(g = 0; g < a->G; ++g)
      a->accEps[iG(n, g)] = 0;
  }

  for(g = 0; g < a->G; ++g){
    a->accPhi[g] = 0;
    a->accAlp[g] = 0;
    a->accDel[g] = 0;
  }
}

__host__ Chain *newChain(Config *cfg){ /* host */
  int n, g, N, G, *grp;
  count_t *y;
  num_t *lqts, s = 0, tmp, *tmpv;
  Chain *host_a, *dev_a;

  y = readData(cfg);
  
  N = cfg->N;
  G = cfg->G;
  
  if(y == NULL)
    return NULL;

  grp = readGrp(cfg);
  
  if(grp == NULL){
    free(y);
    
    return NULL;
  }

  allocChainsDevice(&host_a, &dev_a, cfg);

  /* data and configuration info */

  CUDA_CALL(hipMemcpy(&(host_a->M), &(cfg->M), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->N), &(cfg->N), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->G), &(cfg->G), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->burnin), &(cfg->burnin), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->heterosis), &(cfg->heterosis), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->someParmsFlag), &(cfg->someParmsFlag), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->allParmsFlag), &(cfg->allParmsFlag), sizeof(int), hipMemcpyHostToDevice));  
  
  for(n = 0; n < cfg->N; ++n){
  
    CUDA_CALL(hipMemcpy(&(host_a->grp[n]), &(grp[n]), sizeof(int), hipMemcpyHostToDevice));
    tmp = 0;
    
    for(g = 0; g < cfg->G; ++g){
      CUDA_CALL(hipMemcpy(&(host_a->y[iG(n, g)]), &(y[iG(n, g)]), sizeof(count_t), hipMemcpyHostToDevice));
      tmp += y[iG(n, g)];
    }
    tmp /= cfg->G;
    
    CUDA_CALL(hipMemcpy(&(host_a->yMeanG[n]), &(tmp), sizeof(num_t), hipMemcpyHostToDevice));
  }
    
    
    
  /* initialization constants */
  
  CUDA_CALL(hipMemcpy(&(host_a->sigC0), &(cfg->sigC0), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->d0), &(cfg->d0), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->aTau), &(cfg->aTau), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->aAlp), &(cfg->aAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->aDel), &(cfg->aDel), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->bTau), &(cfg->bTau), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->bAlp), &(cfg->bAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->bDel), &(cfg->bDel), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->gamPhi), &(cfg->gamPhi), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->gamAlp), &(cfg->gamAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->gamDel), &(cfg->gamDel), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->sigPhi0), &(cfg->sigPhi0), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->sigAlp0), &(cfg->sigAlp0), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->sigDel0), &(cfg->sigDel0), sizeof(num_t), hipMemcpyHostToDevice));
  
  /* hyperparameters */
  
  CUDA_CALL(hipMemcpy(&(host_a->sigC[0]), &(cfg->sigC), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->d[0]), &(cfg->d), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->tau[0]), &(cfg->tau), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->thePhi[0]), &(cfg->thePhi), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->theAlp[0]), &(cfg->theAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->theDel[0]), &(cfg->theDel), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->sigPhi[0]), &(cfg->sigPhi), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->sigAlp[0]), &(cfg->sigAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->sigDel[0]), &(cfg->sigDel), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->piAlp[0]), &(cfg->piAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->piDel[0]), &(cfg->piDel), sizeof(num_t), hipMemcpyHostToDevice));
  
  /* choices to hold hyperparameters constant */
  
  CUDA_CALL(hipMemcpy(&(host_a->constSigC), &(cfg->constSigC), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->constD), &(cfg->constD), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->constTau), &(cfg->constTau), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->constThePhi), &(cfg->constThePhi), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->constTheAlp), &(cfg->constTheAlp), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->constTheDel), &(cfg->constTheDel), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->constSigPhi), &(cfg->constSigPhi), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->constSigAlp), &(cfg->constSigAlp), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->constSigDel), &(cfg->constSigDel), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->constPiAlp), &(cfg->constPiAlp), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&(host_a->constPiDel), &(cfg->constPiDel), sizeof(int), hipMemcpyHostToDevice));
  
  lqts = (num_t*) malloc(cfg->N * sizeof(num_t));
  tmpv = (num_t*) malloc(cfg->G * sizeof(num_t));
  
  s = 0;
  for(n = 0; n < cfg->N; ++n){
    for(g = 0; g < cfg->G; ++g)
      tmpv[g] = y[iG(n, g)];
      
    qsort(tmpv, cfg->N, sizeof(num_t), cmpfunc);    
    lqts[n] = log(tmpv[(int) floor(cfg->G * 0.75)]);
    s += lqts[n];
  }
  
  s /= cfg->N;
  
  for(n = 0; n < cfg->N; ++n){
    tmp = lqts[n] - s;
    CUDA_CALL(hipMemcpy(&(host_a->c[iN(0, n)]), &(tmp), sizeof(num_t), hipMemcpyHostToDevice));
  }
  
  newChain_kernel1<<<NBLOCKS, NTHREADS>>>(dev_a);
  newChain_kernel2<<<1, 1>>>(dev_a);
  
  free(lqts);
  free(tmpv);
  free(grp);
  free(y);
    
  return a;
}