#include "hip/hip_runtime.h"
#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

__host__ int cmpfunc (const void *a, const void *b){
   return ( *(num_t*)a - *(num_t*)b );
}

__global__ void curand_setup_kernel(Chain *a, unsigned int seed){
  int G = a->G;
  int id = GENE;
  hiprand_init(seed, id, 0, &(a->states[id]));
}

__global__ void newChain_kernel1(Chain *a){ /* kernel <<<G, 1>>> */
  int n, N = a->N, G = a->G;
  int g = GENE;
  num_t u;

  a->phi[iG(0, g)] = 1; /* rnormal(a->thePhi[0], a->sigPhi[0]);*/

  u = 1; /* runiform(0, 1) */;
  if(u < a->piAlp[0]){
    a->alp[iG(0, g)] = 0;
  } else {
    a->alp[iG(0, g)] = 1; /* rnormal(a->theAlp[0], a->sigAlp[0]); */
  }
    
  u = 1; /* runiform(0, 1); */
  if(u < a->piDel[0]){
    a->del[iG(0, g)] = 0;
  } else {
    a->del[iG(0, g)] = 1; /* rnormal(a->theDel[0], a->sigDel[0]);*/
  }
 
  a->eta[iG(0, g)] = 1; /* 1/sqrt(rgamma(a->d[0] / 2, 
                 a->d[0] * a->tau[0] * a->tau[0] / 2, 0)); */

  for(n = 0; n < a->N; ++n)
    a->eps[iNG(0, n, g)] = 1; /* rnormal(0, a->eta[iG(0, g)]); */
    
}

__global__ void newChain_kernel2(Chain *a){ /* kernel <<<1, 1>>> */
  int n, g, G = a->G;

  a->mC = 0;
  a->mSigC = 0;

  a->mEps = 0;
  a->mEta = 0;
  a->mD = 0;
  a->mTau = 0;

  a->mPhi = 0;
  a->mAlp = 0;
  a->mDel = 0;

  a->mThePhi = 0;
  a->mTheAlp = 0;
  a->mTheDel = 0;

  a->mSigPhi = 0;
  a->mSigAlp = 0;
  a->mSigDel = 0;

  a->mPiAlp = 0;
  a->mPiDel = 0;

  a->tuneD = 100;

  for(n = 0; n < a->N; ++n)
    a->tuneC[n] = 1;

  for(g = 0; g < a->G; ++g){
    a->tunePhi[g] = 1;

    for(n = 0; n < a->N; ++n)
      a->tuneEps[iG(n, g)] = 1;
  }
  
  a->accD = 0;

  for(n = 0; n < a->N; ++n){
    a->accC[n] = 0;
  
    for(g = 0; g < a->G; ++g)
      a->accEps[iG(n, g)] = 0;
  }

  for(g = 0; g < a->G; ++g){
    a->accPhi[g] = 0;
    a->accAlp[g] = 0;
    a->accDel[g] = 0;
  }
}

__host__ void newChain(Chain **host_a, Chain **dev_a, Config *cfg){ /* host */
  int n, g, G, *grp;
  count_t *y;
  num_t *lqts, s = 0, tmp, *tmpv, *yMeanG;

  y = readData(cfg);
  G = cfg->G;
  
  if(y == NULL)
    return;

  grp = readGrp(cfg);
  
  if(grp == NULL){
    free(y);
    return;
  }

  allocChainDevice(host_a, dev_a, cfg);
  
  /* data and configuration info */

  CUDA_CALL(hipMemcpy(&((*dev_a)->M), &(cfg->M), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->N), &(cfg->N), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->G), &(cfg->G), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->burnin), &(cfg->burnin), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->heterosis), &(cfg->heterosis), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->someParmsFlag), &(cfg->someParmsFlag), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->allParmsFlag), &(cfg->allParmsFlag), sizeof(int), hipMemcpyHostToDevice));  
  
  /* initialization constants */
  
  CUDA_CALL(hipMemcpy(&((*dev_a)->sigC0), &(cfg->sigC0), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->d0), &(cfg->d0), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->aTau), &(cfg->aTau), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->aAlp), &(cfg->aAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->aDel), &(cfg->aDel), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->bTau), &(cfg->bTau), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->bAlp), &(cfg->bAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->bDel), &(cfg->bDel), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->gamPhi), &(cfg->gamPhi), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->gamAlp), &(cfg->gamAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->gamDel), &(cfg->gamDel), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->sigPhi0), &(cfg->sigPhi0), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->sigAlp0), &(cfg->sigAlp0), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->sigDel0), &(cfg->sigDel0), sizeof(num_t), hipMemcpyHostToDevice));
  
  /* choices to hold hyperparameters constant */
  
  CUDA_CALL(hipMemcpy(&((*dev_a)->constSigC), &(cfg->constSigC), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->constD), &(cfg->constD), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->constTau), &(cfg->constTau), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->constThePhi), &(cfg->constThePhi), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->constTheAlp), &(cfg->constTheAlp), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->constTheDel), &(cfg->constTheDel), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->constSigPhi), &(cfg->constSigPhi), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->constSigAlp), &(cfg->constSigAlp), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->constSigDel), &(cfg->constSigDel), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->constPiAlp), &(cfg->constPiAlp), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(&((*dev_a)->constPiDel), &(cfg->constPiDel), sizeof(int), hipMemcpyHostToDevice));

  /* hyperparameters */
  
  CUDA_CALL(hipMemcpy((*host_a)->sigC, &(cfg->sigC), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy((*host_a)->d, &(cfg->d), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy((*host_a)->tau, &(cfg->tau), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy((*host_a)->thePhi, &(cfg->thePhi), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy((*host_a)->theAlp, &(cfg->theAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy((*host_a)->theDel, &(cfg->theDel), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy((*host_a)->sigPhi, &(cfg->sigPhi), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy((*host_a)->sigAlp, &(cfg->sigAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy((*host_a)->sigDel, &(cfg->sigDel), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy((*host_a)->piAlp, &(cfg->piAlp), sizeof(num_t), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy((*host_a)->piDel, &(cfg->piDel), sizeof(num_t), hipMemcpyHostToDevice));
  

  /* data */
  
  CUDA_CALL(hipMemcpy((*host_a)->grp, grp, cfg->N * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy((*host_a)->y, y, cfg->N * cfg->G * sizeof(int), hipMemcpyHostToDevice));
  
  yMeanG = (num_t*) malloc(cfg->N * sizeof(num_t));
   
  for(n = 0; n < cfg->N; ++n){
    tmp = 0;
    
    for(g = 0; g < cfg->G; ++g)
      tmp += y[iG(n, g)];
    
    tmp /= cfg->G;
    yMeanG[n] = tmp;   
  }

  CUDA_CALL(hipMemcpy((*host_a)->yMeanG, yMeanG, cfg->N * sizeof(num_t), hipMemcpyHostToDevice));
  
  lqts = (num_t*) malloc(cfg->N * sizeof(num_t));
  tmpv = (num_t*) malloc(cfg->G * sizeof(num_t));
  
  s = 0;
  for(n = 0; n < cfg->N; ++n){
    for(g = 0; g < cfg->G; ++g)
      tmpv[g] = y[iG(n, g)];
      
    qsort(tmpv, cfg->N, sizeof(num_t), cmpfunc);    
    lqts[n] = log(tmpv[(int) floor(cfg->G * 0.75)]);
    s += lqts[n];
  }
  
  s /= cfg->N;
  
  for(n = 0; n < cfg->N; ++n)
    tmpv[n] = lqts[n] - s;
  
  CUDA_CALL(hipMemcpy((*host_a)->c, tmpv, cfg->N *sizeof(num_t), hipMemcpyHostToDevice));
  
  /* set up hiprand states */
  
  curand_setup_kernel<<<NBLOCKS, NTHREADS>>>(*dev_a, cfg->seed);
  
  newChain_kernel1<<<NBLOCKS, NTHREADS>>>(*dev_a);
  newChain_kernel2<<<1, 1>>>(*dev_a);
  
  free(yMeanG);
  free(lqts);
  free(tmpv);
  free(grp);
  free(y);
}