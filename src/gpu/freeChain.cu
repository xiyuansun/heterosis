#include <Chain.h>
#include <Config.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <functions.h>
#include <stdio.h>
#include <stdlib.h>

__host__ void freeChain(Chain *a, Config *cfg){
  
  Chain *host_a = (Chain*) malloc(sizeof(Chain));
  CUDA_CALL(hipMemcpy(host_a, a, sizeof(Chain), hipMemcpyDeviceToHost));
  CUDA_CALL(hipFree(a));
  
  /* data */  

  CUDA_CALL(hipFree(host_a->y));
  CUDA_CALL(hipFree(host_a->yMeanG));
  CUDA_CALL(hipFree(host_a->grp));

  /* parameters */
  
  CUDA_CALL(hipFree(host_a->c));
  CUDA_CALL(hipFree(host_a->sigC));
  CUDA_CALL(hipFree(host_a->eps));
  CUDA_CALL(hipFree(host_a->eta));
  CUDA_CALL(hipFree(host_a->d));
  CUDA_CALL(hipFree(host_a->tau));
  CUDA_CALL(hipFree(host_a->phi));
  CUDA_CALL(hipFree(host_a->thePhi));
  CUDA_CALL(hipFree(host_a->sigPhi));
  CUDA_CALL(hipFree(host_a->alp));
  CUDA_CALL(hipFree(host_a->theAlp));
  CUDA_CALL(hipFree(host_a->sigAlp));
  CUDA_CALL(hipFree(host_a->piAlp));
  CUDA_CALL(hipFree(host_a->del));
  CUDA_CALL(hipFree(host_a->theDel));
  CUDA_CALL(hipFree(host_a->sigDel));
  CUDA_CALL(hipFree(host_a->piDel));
  
  /* temporary and return values */
  
  CUDA_CALL(hipFree(host_a->tmp1));
  CUDA_CALL(hipFree(host_a->tmp2));

  CUDA_CALL(hipFree(host_a->Old));
  CUDA_CALL(hipFree(host_a->New));
  CUDA_CALL(hipFree(host_a->lOld));
  CUDA_CALL(hipFree(host_a->lNew));
  
  /* tuning parameters for Metropolis steps */
  
  CUDA_CALL(hipFree(host_a->tuneC));
  CUDA_CALL(hipFree(host_a->tunePhi));
  CUDA_CALL(hipFree(host_a->tuneEps));

  /* number of acceptances for Metropolis steps */

  CUDA_CALL(hipFree(host_a->accC));
  CUDA_CALL(hipFree(host_a->accPhi));
  CUDA_CALL(hipFree(host_a->accAlp));
  CUDA_CALL(hipFree(host_a->accDel));
  CUDA_CALL(hipFree(host_a->accEps));
  
  free(host_a);
}