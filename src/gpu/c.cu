#include "hip/hip_runtime.h"
#include <Chain.h>
#include <constants.h>
#include <constants.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <deviceFunctions.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/reduce.h>

__global__ void lC_kernel1(Chain *a, int n){ /* kernel <<<G, 1>>> */
  int g = GENE, N = a->N, G = a->G;
  
  if(g < G)
    a->tmp1[g] = exp(a->eps[iNG(a->mEps, n, g)] + mu(a, n, a->phi[iG(a->mPhi, g)], 
                    a->alp[iG(a->mAlp, g)], a->del[iG(a->mDel, g)]));
}

__global__ void lC_kernel2(Chain *a, int n, int newArg){ /* kernel <<<1, 1>>> */
  num_t arg, ret;

  if(newArg){
    arg = a->New[n];
  } else {
    arg = a->Old[n];
  }

  ret = arg * a->G * a->yMeanG[n] - exp(arg) * a->s1 - (arg*arg) / 
        (2 * a->sigC[a->mSigC] * a->sigC[a->mSigC]);

  if(newArg){
    a->lNew[n] = ret;
  } else {
    a->lOld[n] = ret;
  }
}

__host__ void lC(Chain *host_a, Chain *dev_a, Config *cfg, int n, int newArg){ /* host */
  lC_kernel1<<<NBLOCKS, NTHREADS>>>(dev_a, n);
  
  thrust::device_ptr<num_t> tmp1(host_a->tmp1);  
  num_t s1 = thrust::reduce(tmp1, tmp1 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s1), &s1, sizeof(num_t), hipMemcpyHostToDevice));
  
    printf("thrust c s1 = %0.3f\n", s1);
  
  lC_kernel2<<<1, 1>>>(dev_a, n, newArg);
}

__global__ void sampleC_kernel1(Chain *a){ /* kernel <<<1, N>>> */
  int n = ((blockDim.x * blockIdx.x) + threadIdx.x), N = a->N;
  
  if(n < N){
    a->Old[n] = a->c[iN(a->mC, n)];
    a->New[n] = rnormalDevice(a, n, a->Old[n], a->tuneC[n]);
    
  }
}

__global__ void sampleC_kernel2(Chain *a){ /* kernel <<<1, N>>> */
  int n = ((blockDim.x * blockIdx.x) + threadIdx.x), N = a->N;
  num_t dl, lp, lu;

  if(n < N){ 

    dl = a->lNew[n] - a->lOld[n];
    lp = 0 < dl ? 0 : dl;
    lu = log(runiformDevice(a, 1, 0, 1));
      
    if(lu < lp){ /* accept */
      a->c[iN(a->mC + 1, n)] = a->New[n];
      a->tuneC[n] *= 1.1; /* Increase the proposal variance to avoid  
                                       gettiG stuck in a mode */
                                       
      if(a->mC >= a->burnin)                                 
        ++a->accC[n];
    } else { /* reject */
      a->c[iN(a->mC + 1, n)] = a->Old[n];
      a->tuneC[n] /= 1.1; /* If you're rejectiG too often, decrease the  
                                       proposal variance to sample closer to 
                                       the last accepted value. */
    }
  }
}
 
__global__ void sampleC_kernel3(Chain *a){ /* kernel <<<1, 1>>> */
  ++a->mC;
}

__host__ void sampleC(Chain *host_a, Chain *dev_a, Config *cfg){ /* host */
  int n, N = cfg->N;
  int nthreads = (N < MAXTHREADS ? N : MAXTHREADS);
  int nblocks = ceil(cfg->N / NTHREADS) + 1;
  
  sampleC_kernel1<<<nblocks, nthreads>>>(dev_a);

  for(n = 0; n < cfg->N; ++n){ 
    lC(host_a, dev_a, cfg, n, 1);
    lC(host_a, dev_a, cfg, n, 0);
  }

  sampleC_kernel2<<<nblocks, nthreads>>>(dev_a);
  sampleC_kernel3<<<1, 1>>>(dev_a);
}