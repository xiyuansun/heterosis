#include "hip/hip_runtime.h"
#include <Chain.h>
#include <Config.h>
#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

void printHyper(Chain *a, Config *cfg){

  int m;
  num_t *sigC, *d, *tau, *thePhi, *theAlp, *theDel, *sigPhi, *sigAlp, *sigDel, *piAlp, *piDel;
  char file[BUF];
  FILE *fp;
  double time;
  clock_t start = clock();
  
  if(cfg->hyperFlag){
    fprintf(cfg->log, "  Printing hyperparameters.\n");

    sprintf(file, "../out/hyper/chain%d.txt", cfg->chainNum);
    fp = fopen(file, "w");
    
    if(fp == NULL){
      printf("ERROR: unable to create file, %s\n", file);
      return;
    }
    
    sigC   = (num_t*) malloc((cfg->M + 1) * sizeof(num_t));
    d      = (num_t*) malloc((cfg->M + 1) * sizeof(num_t));
    tau    = (num_t*) malloc((cfg->M + 1) * sizeof(num_t));
    thePhi = (num_t*) malloc((cfg->M + 1) * sizeof(num_t));
    theAlp = (num_t*) malloc((cfg->M + 1) * sizeof(num_t));
    theDel = (num_t*) malloc((cfg->M + 1) * sizeof(num_t));
    sigPhi = (num_t*) malloc((cfg->M + 1) * sizeof(num_t));
    sigAlp = (num_t*) malloc((cfg->M + 1) * sizeof(num_t));
    sigDel = (num_t*) malloc((cfg->M + 1) * sizeof(num_t));
    piAlp  = (num_t*) malloc((cfg->M + 1) * sizeof(num_t));
    piDel  = (num_t*) malloc((cfg->M + 1) * sizeof(num_t));
    
    CUDA_CALL(hipMemcpy(sigC,   a->sigC,   (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(d,      a->d,      (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(tau,    a->tau,    (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(thePhi, a->thePhi, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(theAlp, a->theAlp, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(theDel, a->theDel, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(sigPhi, a->sigPhi, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(sigAlp, a->sigAlp, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(sigDel, a->sigDel, (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(piAlp,  a->piAlp,  (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(piDel,  a->piDel,  (cfg->M + 1) * sizeof(num_t), hipMemcpyDeviceToHost));
    
    fprintf(fp, "sigC d tau thePhi theAlp theDel sigPhi sigAlp sigDel piAlp piDel\n");
    
    for(m = 0; m <= cfg->M; ++m){ 
      fprintf(fp, NUM_TF, sigC[m]);   fprintf(fp, " ");   
      fprintf(fp, NUM_TF, d[m]);      fprintf(fp, " "); 
      fprintf(fp, NUM_TF, tau[m]);    fprintf(fp, " "); 
      fprintf(fp, NUM_TF, thePhi[m]); fprintf(fp, " ");  
      fprintf(fp, NUM_TF, theAlp[m]); fprintf(fp, " "); 
      fprintf(fp, NUM_TF, theDel[m]); fprintf(fp, " "); 
      fprintf(fp, NUM_TF, sigPhi[m]); fprintf(fp, " "); 
      fprintf(fp, NUM_TF, sigAlp[m]); fprintf(fp, " "); 
      fprintf(fp, NUM_TF, sigDel[m]); fprintf(fp, " "); 
      fprintf(fp, NUM_TF, piAlp[m]);  fprintf(fp, " "); 
      fprintf(fp, NUM_TF, piDel[m]);  fprintf(fp, " "); 
      
      fprintf(fp, "\n");
    }
    
    free(sigC);
    free(d);
    free(tau);
    free(thePhi);
    free(theAlp);
    free(theDel);
    free(sigPhi);
    free(sigAlp);
    free(sigDel);
    free(piAlp);
    free(piDel);
    
    fclose(fp);
  }
  
  time = ((double) clock() - start) / (60 * CLOCKS_PER_SEC);
  fprintf(cfg->time, "%0.3f ", time);
}