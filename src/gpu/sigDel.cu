#include "hip/hip_runtime.h"
#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/reduce.h>

__global__ void sampleSigDel_kernel1(Chain *a){ /* kernel <<<G, 1>>> */
  int g = IDX, G = a->G;

  for(g = 0; g < a->G; ++g){ 
    if(pow(a->del[iG(a->mDel, g)], 2) > 1e-6){
      a->tmp1[g] = pow(a->del[iG(a->mDel, g)] - a->theDel[a->mTheDel], 2);
      a->tmp2[g] = 1;
    } else {
      a->tmp1[g] = 0;
      a->tmp2[g] = 0;
    }
  } 
}

__global__ void sampleSigDel_kernel2(Chain *a){ /* kernel <<<1, 1>>> */
  num_t shape = (a->s2 - 1) / 2;
  num_t rate = a->s1 / 2;
  num_t lb = 1/pow(a->sigDel0, 2);

  if(shape >= 1 && rate > 0){
    a->sigDel[a->mSigDel + 1] = 1/sqrt(rgammaDevice(a, 1, shape, rate, lb));
  } else {
    a->sigDel[a->mSigDel + 1] = a->sigDel[a->mSigDel];
  }

  ++a->mSigDel;
}

__host__ void sampleSigDel(Chain *host_a, Chain *dev_a, Config *cfg){ /* host */
  float myTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  fprintf(cfg->log, "sigDel ");

  if(cfg->constSigDel || !cfg->heterosis)
    return;

  sampleSigDel_kernel1<<<G_GRID, G_BLOCK>>>(dev_a);
  
  thrust::device_ptr<num_t> tmp1(host_a->tmp1);  
  num_t s1 = thrust::reduce(tmp1, tmp1 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s1), &s1, sizeof(num_t), hipMemcpyHostToDevice));
  
  thrust::device_ptr<num_t> tmp2(host_a->tmp2);  
  num_t s2 = thrust::reduce(tmp2, tmp2 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s2), &s2, sizeof(num_t), hipMemcpyHostToDevice));
 
  sampleSigDel_kernel2<<<1, 1>>>(dev_a);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&myTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  fprintf(cfg->time, "%0.3f ", myTime/MILLISECS); /* elapsed time */
  hipDeviceSynchronize();
}