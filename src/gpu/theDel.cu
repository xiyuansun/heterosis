#include "hip/hip_runtime.h"
#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/reduce.h>

__global__ void sampleTheDel_kernel1(Chain *a){ /* kernel <<<G, 1>>> */
  int g = IDX, G = a->G;

  if(g < G){ 
    if(pow(a->del[iG(a->mDel, g)], 2) > 1e-6){
      a->tmp1[g] = 1;
      a->tmp2[g] = a->del[iG(a->mDel, g)];
    } else {
      a->tmp1[g] = 0;
      a->tmp2[g] = 0;
    }
  }
} 

__global__ void sampleTheDel_kernel2(Chain *a){ /* kernel <<<1, 1>>> */

  num_t gs = pow(a->gamDel, 2);
  num_t ss = pow(a->sigDel[a->mSigDel], 2);
  num_t den = a->s1 * gs + ss;

  num_t m = gs * a->s2 / den;
  num_t s = sqrt(gs * ss / den);

  a->theDel[a->mTheDel + 1] = rnormalDevice(a, 1, m, s);
  ++a->mTheDel;
}

void sampleTheDel(Chain *host_a, Chain *dev_a, Config *cfg){ /* host */

  float myTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  if(cfg->verbose)
    printf("theDel ");

  if(cfg->constTheDel || !cfg->heterosis)
    return;

  sampleTheDel_kernel1<<<G_GRID, G_BLOCK>>>(dev_a);
  
  thrust::device_ptr<num_t> tmp1(host_a->tmp1);  
  num_t s1 = thrust::reduce(tmp1, tmp1 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s1), &s1, sizeof(num_t), hipMemcpyHostToDevice));
  
  thrust::device_ptr<num_t> tmp2(host_a->tmp2);  
  num_t s2 = thrust::reduce(tmp2, tmp2 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s2), &s2, sizeof(num_t), hipMemcpyHostToDevice));
  
  sampleTheDel_kernel2<<<1, 1>>>(dev_a);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&myTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  fprintf(cfg->time, "%0.3f ", myTime/MILLISECS); /* elapsed time */
}