#include "hip/hip_runtime.h"
#include <Chain.h>
#include <constants.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

__device__ num_t lEps(Chain *a, int n, int g, num_t arg){ /* device */
  int N = a->N, G = a->G;
  return a->y[iG(n, g)] * arg - exp(a->c[iN(a->mC, n)] + arg + mu(a, n, a->phi[iG(a->mPhi, g)], 
                                     a->alp[iG(a->mAlp, g)], a->del[iG(a->mDel, g)])) 
                          - (arg * arg) / (2 * pow(a->eta[iG(a->mEta, g)], 2));
}

__global__ void sampleEps_kernel1(Chain *a){ /* kernel <<<G, 1>>> */
  int n, g = IDX, N = a->N, G = a->G;
  num_t old, nw, dl, lp, lu;

  if(g < G){
    for(n = 0; n < N; ++n){ 
      old = a->eps[iNG(a->mEps, n, g)];
      nw = rnormalDevice(a, g, old, a->tuneEps[iG(n, g)]);

      dl = lEps(a, n, g, nw) - lEps(a, n, g, old);
      lp = 0 < dl ? 0 : dl;
      lu = log(runiformDevice(a, g, 0, 1));
      
      if(lu < lp){ /* accept */
        a->eps[iNG(a->mEps + 1, n, g)] = nw;
        a->tuneEps[iG(n, g)] *= 1.1;
        
        if(a->mEps >= a->burnin)
          ++a->accEps[iG(n, g)]; 
      } else { /* reject */
        a->eps[iNG(a->mEps + 1, n, g)] = old;
        a->tuneEps[iG(n, g)] /= 1.1;
      }
    }
  }
}

__global__ void sampleEps_kernel2(Chain *a){ /* kernel <<<1, 1>>> */
  ++a->mEps;
}

void sampleEps(Chain *host_a, Chain *dev_a, Config *cfg){ /* host */
  float myTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  fprintf(cfg->log, "eps ");

  sampleEps_kernel1<<<G_GRID, G_BLOCK>>>(dev_a);
  sampleEps_kernel2<<<1, 1>>>(dev_a);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&myTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  fprintf(cfg->time, "%0.3f ", myTime/MILLISECS); /* elapsed time */
  hipDeviceSynchronize();
}