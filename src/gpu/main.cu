#include <Chain.h>
#include <Config.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <constants.h>
#include <functions.h>
#include <stdlib.h>
#include <stdio.h>

int main(int argc, char **argv){
 /* oneChain(argc, argv);*/
 
  Config *cfg = config(argc, argv);  printf("1\n");
  Chain *a = allocChain(cfg); printf("2\n");
  printChain(a, cfg, 0); printf("3\n");
  freeChain(a, cfg, 0);printf("4\n");
 
  return EXIT_SUCCESS;
}