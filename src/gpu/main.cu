#include <Chain.h>
#include <Config.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <constants.h>
#include <functions.h>
#include <stdlib.h>
#include <stdio.h>

int main(int argc, char **argv){
  
  Config *cfg = config(argc, argv); 
  Chain *host_a, *dev_a;
  
  newChain(&host_a, &dev_a, cfg); 
  printChain(host_a, dev_a, cfg);
  
  freeChain(host_a, cfg, 0); 
  hipFree(dev_a);
 
  return EXIT_SUCCESS;
}