#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <functions.h>
#include <stdlib.h>
#include <stdio.h>

void mcmc(int argc, char **argv){
  int i;
  Config *cfg = config(argc, argv); 
  Chain *host_a = NULL, *dev_a = NULL;
  
  if(cfg->verbose)
    printf("Running %d chain(s).\n", cfg->chains);

  newChain(&host_a, &dev_a, cfg);  
  
  if(host_a == NULL){
    free(cfg);
    exit(EXIT_FAILURE);
  }
  
  for(i = 0; i < cfg->chains; ++i){
  
    if(cfg->verbose)
      printf("  Chain %d\n", i);

    cfg->chainNum = i;
    
    printChain(host_a, dev_a, cfg);
    
    runChain(host_a, dev_a, cfg);
    summarizeChain(host_a, dev_a, cfg);

    resetChain(host_a, dev_a, cfg);
  }
  
  freeChain(host_a, cfg, 0);
  hipFree(dev_a);
  
  if(cfg->verbose)
    printf("Done running mcmc.\n");

  free(cfg);
}