#include "hip/hip_runtime.h"
#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/reduce.h>

__global__ void lD_kernel1(Chain *a, int newArg){ /* kernel <<<1, 1>>> */
  if(newArg){
    if(a->New[0] <= 0 || a->New[0] > a->d0)
     a->lNew[0] = NUM_TMIN;
  } else {
    if(a->Old[0] <= 0 || a->Old[0] > a->d0)
      a->lOld[0] = NUM_TMIN; 
  }
}

__global__ void lD_kernel2(Chain *a){ /* kernel <<<G, 1>>> */
  int g = IDX, G = a->G;

  if(g < G){ 
    a->tmp1[g] = 2 * log(a->eta[iG(a->mEta, g)]);
    a->tmp2[g] = 1/(a->eta[iG(a->mEta, g)] * a->eta[iG(a->mEta, g)]);
  }
}

__global__ void lD_kernel3(Chain *a, int newArg){ /* kernel <<<1, 1>>> */
  num_t arg, ret, tmp;
 
  if(newArg){ 
    arg = a->New[0];
  } else{
    arg = a->Old[0];
  }

  tmp = arg * a->tau[a->mTau] * a->tau[a->mTau] / 2;
  ret = -a->G * lgamma(arg/2) + (a->G * arg / 2) * log(tmp);
  ret -= (arg/2 + 1) * a->s1 - tmp * a->s2;

  if(newArg){
    a->lNew[0] = ret;
  } else{
    a->lOld[0] = ret;
  }
}

__host__ void lD(Chain *host_a, Chain *dev_a, Config *cfg, int newArg){ /* host */
  
  lD_kernel1<<<1, 1>>>(dev_a, newArg);
  lD_kernel2<<<G_GRID, G_BLOCK>>>(dev_a);
  
  thrust::device_ptr<num_t> tmp1(host_a->tmp1);  
  num_t s1 = thrust::reduce(tmp1, tmp1 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s1), &s1, sizeof(num_t), hipMemcpyHostToDevice));
  
  thrust::device_ptr<num_t> tmp2(host_a->tmp2);  
  num_t s2 = thrust::reduce(tmp2, tmp2 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s2), &s2, sizeof(num_t), hipMemcpyHostToDevice));
  
  lD_kernel3<<<1, 1>>>(dev_a, newArg);
}

__global__ void sampleD_kernel1(Chain *a){ /* kernel <<<1, 1>>> */
  a->Old[0] = a->d[a->mD];
  
  do {
    a->New[0] = rnormalDevice(a, 1, a->Old[0], a->tuneD);
  } while(a->New[0] < 0);
}

__global__ void sampleD_kernel2(Chain *a){ /* kernel <<<1, 1>>> */
  num_t dl = a->lNew[0] - a->lOld[0];
  num_t lp = 0 < dl ? 0 : dl;
  num_t lu = log(runiformDevice(a, 1, 0, 1));

  if(lu < lp){ /* accept */
    a->d[a->mD + 1] = a->New[0];
    a->tuneD *= 1.1; /* Increase the proposal variance to avoid getting 
                                  stuck in a mode */
    if(a->mD >= a->burnin) 
      ++a->accD;
  } else { /* reject */
    a->d[a->mD + 1] = a->Old[0];
    a->tuneD /= 1.1; /* If you're rejecting too often, decrease the proposal 
                                  variance to sample closer to the last accepted value. */
  }

  ++a->mD;
}

__host__ void sampleD(Chain *host_a, Chain *dev_a, Config *cfg){ /* host */

  float myTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  fprintf(cfg->log, "d ");

  if(cfg->constD)
    return;
   
  sampleD_kernel1<<<1, 1>>>(dev_a);

  lD(host_a, dev_a, cfg, 1);
  lD(host_a, dev_a, cfg, 0);

  sampleD_kernel2<<<1, 1>>>(dev_a);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&myTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  fprintf(cfg->time, "%0.3f ", myTime/MILLISECS); /* elapsed time */
}
