#include "hip/hip_runtime.h"
#include <Chain.h>
#include <constants.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

void sampleEta_kernel1(Chain *a){ /* kernel <<<1, 1>>> */
  a->s1 = (a->N + a->d[a->mD]) / 2; 
}

void sampleEta_kernel2(Chain *a){ /* kernel <<<G, 1>>> */
  int n, g;
  int M = a->M, N = a->N, G = a->G;
  
  num_t shape = a->s1, rate;

  for(g = 0; g < G; ++g){

    rate = 0;
    for(n = 0; n < N; ++n) 
      rate += a->eps[a->mEps][n][g] * a->eps[a->mEps][n][g];
  
    rate = (rate + a->d[a->mD] * a->tau[a->mTau] * a->tau[a->mTau]) / 2; 

    if(shape >= 1 && rate > 0){
      a->eta[a->mEta + 1][g] = 1/sqrt(rgamma(shape, rate, 0));
    } else {
      a->eta[a->mEta + 1][g] = a->eta[a->mEta][g];
    }
  }
}

void sampleEta_kernel3(Chain *a){ /* kernel <<<1, 1>>> */
  ++a->mEta;
}

void sampleEta(Chain *a){
  sampleEta_kernel1(a);
  sampleEta_kernel2(a);
  sampleEta_kernel3(a);
}