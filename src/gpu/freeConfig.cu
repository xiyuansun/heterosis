#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <functions.h>
#include <stdio.h>
#include <stdlib.h>

__host__ void freeConfig(Config *cfg){
  free(cfg->y);
  free(cfg->grp);
  free(cfg->yMeanG);
  
  hipFree(cfg->devY);
  hipFree(cfg->devGrp);
  hipFree(cfg->devYMeanG);

  free(cfg);  
}