#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <functions.h>
#include <stdio.h>
#include <stdlib.h>

__host__ void allocConfig(Config *cfg){
  free(cfg->y);
  free(cfg->group);
  free(cfg->yMeanG);
  
  hipFree(cfg->devY);
  hipFree(cfg->devGroup);
  hipFree(cfg->devYMeanG);

  free(cfg);  
}