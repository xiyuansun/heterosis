#include "hip/hip_runtime.h"
#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/reduce.h>

__global__ void samplePiAlp_kernel1(Chain *a){ /* kernel <<<G, 1>>> */
  int g = IDX, G = a->G;

  if(g < G){ 
    if(pow(a->alp[iG(a->mAlp, g)], 2) > 1e-6){
      a->tmp1[g] = 1;
    } else {
      a->tmp1[g] = 0;
    }
  }
}

__global__ void samplePiAlp_kernel2(Chain *a){ /* kernel <<<1, 1>>> */
  a->piAlp[a->mPiAlp + 1] = rbetaDevice(a, 1, a->G + a->s1 + a->aTau, a->s1 + a->bTau);
  ++a->mPiAlp;
} 

__host__ void samplePiAlp(Chain *host_a, Chain *dev_a, Config *cfg){ /* host */

  float myTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  if(cfg->verbose)
    printf("piAlp ");

  if(cfg->constPiAlp)
    return;

  samplePiAlp_kernel1<<<G_GRID, G_BLOCK>>>(dev_a);
  
  thrust::device_ptr<num_t> tmp1(host_a->tmp1);  
  num_t s1 = thrust::reduce(tmp1, tmp1 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s1), &s1, sizeof(num_t), hipMemcpyHostToDevice));
  
  samplePiAlp_kernel2<<<1, 1>>>(dev_a);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&myTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  fprintf(cfg->time, "%0.3f ", myTime/MILLISECS); /* elapsed time */
}