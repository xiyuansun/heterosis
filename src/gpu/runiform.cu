#include <Chain.h>
#include <constants.h>
#include <hiprand/hiprand_kernel.h>
#include <functions.h>
#include <math.h>
#include <stdlib.h>

__host__ num_t runiform(num_t lb, num_t ub){
  num_t u = ((num_t) rand()) / ((num_t) RAND_MAX);
  return (ub - lb) * u + lb;
}

/* See deviceFunctions.h for device version. */