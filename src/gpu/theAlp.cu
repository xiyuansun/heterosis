#include "hip/hip_runtime.h"
#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/reduce.h>

__global__ void sampleTheAlp_kernel1(Chain *a){ /* kernel <<<G, 1>>> */
  int g = IDX, G = a->G;

  if(g < G){
    if(pow(a->alp[iG(a->mAlp, g)], 2) > 1e-6){
      a->tmp1[g] = 1;
      a->tmp2[g] = a->alp[iG(a->mAlp, g)];
    } else {
      a->tmp1[g] = 0;
      a->tmp2[g] = 0;
    }
  }
} 

__global__ void sampleTheAlp_kernel2(Chain *a){ /* kernel <<<1, 1>>> */

  num_t gs = a->gamAlp * a->gamAlp;
  num_t ss = a->sigAlp[a->mSigAlp] * a->sigAlp[a->mSigAlp];
  num_t den = a->s1 * gs + ss;

  num_t m = gs * a->s2 / den;
  num_t s = sqrt(gs * ss / den);

  a->theAlp[a->mTheAlp + 1] = rnormalDevice(a, 1, m, s);
  ++a->mTheAlp;
}

__host__ void sampleTheAlp(Chain *host_a, Chain *dev_a, Config *cfg){ /* host */
  fprintf(cfg->log, "theAlp ");

  if(cfg->constTheAlp)
    return;

  sampleTheAlp_kernel1<<<G_GRID, G_BLOCK>>>(dev_a);
  
  thrust::device_ptr<num_t> tmp1(host_a->tmp1);  
  num_t s1 = thrust::reduce(tmp1, tmp1 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s1), &s1, sizeof(num_t), hipMemcpyHostToDevice));
  
  thrust::device_ptr<num_t> tmp2(host_a->tmp2);  
  num_t s2 = thrust::reduce(tmp2, tmp2 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s2), &s2, sizeof(num_t), hipMemcpyHostToDevice));
  
  sampleTheAlp_kernel2<<<G_GRID, G_BLOCK>>>(dev_a);
}
