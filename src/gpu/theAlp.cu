#include "hip/hip_runtime.h"
#include <Chain.h>
#include <Config.h>
#include <constants.h>
#include <functions.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/reduce.h>

__global__ void sampleTheAlp_kernel1(Chain *a){ /* kernel <<<G, 1>>> */
  int g = IDX;

  if(g < a->G){
    if(pow((float) a->alp[g], 2) > 1e-6){
      a->tmp1[g] = 1;
      a->tmp2[g] = a->alp[g];
    } else {
      a->tmp1[g] = 0;
      a->tmp2[g] = 0;
    }
  }
}

__global__ void sampleTheAlp_kernel2(Chain *a){ /* kernel <<<1, 1>>> */

  num_t gs = pow((float) a->gamAlp, 2);
  num_t ss = pow((float) a->sigAlp, 2);
  num_t den = a->s1 * gs + ss;

  num_t m = gs * a->s2 / den;
  num_t s = sqrt(gs * ss / den);

  a->theAlp = rnormalDevice(a, 1, m, s);
}

__host__ void sampleTheAlp(Chain *host_a, Chain *dev_a, Config *cfg){ /* host */

  num_t myTime;
  hipEvent_t start, stop;
  
  if(cfg->constTheAlp)
    return;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  if(cfg->verbose)
    printf("theAlp ");

  sampleTheAlp_kernel1<<<G_GRID, G_BLOCK>>>(dev_a);
 
  thrust::device_ptr<num_t> tmp1(host_a->tmp1);  
  num_t s1 = thrust::reduce(tmp1, tmp1 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s1), &s1, sizeof(num_t), hipMemcpyHostToDevice));
  
  thrust::device_ptr<num_t> tmp2(host_a->tmp2);  
  num_t s2 = thrust::reduce(tmp2, tmp2 + cfg->G);
  CUDA_CALL(hipMemcpy(&(dev_a->s2), &s2, sizeof(num_t), hipMemcpyHostToDevice));
  
  sampleTheAlp_kernel2<<<1, 1>>>(dev_a);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&myTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  cfg->timeTheAlp = myTime / MILLISECS;
}